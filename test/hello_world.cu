
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void helloWorldKernel() {
    printf("Hello, World from GPU!\n");
}

int main() {
    // Launch the kernel with 1 block and 1 thread
    helloWorldKernel<<<1, 1>>>();

    // Wait for GPU to finish before accessing results
    hipDeviceSynchronize();

    std::cout << "Hello, World from CPU!" << std::endl;

    return 0;
}